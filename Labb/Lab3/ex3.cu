#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>
#include <jetson-utils/videoOutput.h>

__global__ void rgb2grayKernel(uchar4 *image, uchar4 *outputImage, int width, int height)
{

    int total = width * height;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < total; i += stride)
    {

        unsigned char gray = image[i].x * 0.299 + image[i].y * 0.587 + image[i].z * 0.114;
        outputImage[i].x = gray;
        outputImage[i].y = gray;
        outputImage[i].z = gray;
    }
}

int main(int argc, char **argv)
{
    // create input/output streams
    videoSource *input = videoSource::Create(argc, argv, ARG_POSITION(0));
    videoOutput *output = videoOutput::Create(argc, argv, ARG_POSITION(1));
    videoOutput *output2 = videoOutput::Create(argc, argv, ARG_POSITION(1));
    uchar4 *outputImage = NULL;
    hipMalloc(&outputImage, sizeof(uchar4) * 720 * 1280);
    if (!input)
        return 0;

    // capture/display loop
    while (true)
    {

        uchar4 *image = NULL;
        //  can be uchar3, uchar4, float3, float4
        int status = 0;                             // see videoSource::Status (OK, TIMEOUT, EOS, ERROR)
        if (!input->Capture(&image, 1000, &status)) // 1000ms timeout (default)
        {
            if (status == videoSource::TIMEOUT)
                continue;
            break; // EOS
        }
        rgb2grayKernel<<<16, 1024>>>(image, outputImage, input->GetWidth(), input->GetHeight());
        if (output != NULL)
        {

            output->Render(image, input->GetWidth(), input->GetHeight());

            // Update status bar
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
                    input->GetHeight(), output->GetFrameRate());
            output->SetStatus(str);

            if (!output->IsStreaming()) // check if the user quit
                break;
        }
        if (output2 != NULL)
        {

            output2->Render(outputImage, input->GetWidth(), input->GetHeight());
            // Update status bar
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
                    input->GetHeight(), output2->GetFrameRate());

            output2->SetStatus(str);
            if (!output2->IsStreaming()) // check if the user quit
                break;
        }
    }
    hipFree(outputImage);
}