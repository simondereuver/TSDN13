#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>
#include <jetson-utils/videoOutput.h>

__global__ void plotHistogramKernel(uchar4 *image, int *histogram, int width, int height, int max_freq)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    uchar4 white_pixel = make_uchar4(255, 255, 255, 255);
    // uchar4 black_pixel = make_uchar4(0, 0, 0, 255);
    if (index < 256)
    {
        int freq = histogram[index] * 256 / max_freq;
        for (int i = 0; i < 256; i++)
        {
            int row = height - i - 1;
            if (i <= freq)
            {
                image[row * width + 2 * index] = white_pixel;
                image[row * width + 2 * index + 1] = white_pixel;
            }
            else
            {
                uchar4 transparent_pixel = make_uchar4(image[row * width + 2 * index].x * 0.7, image[row * width + 2 * index].y * 0.7,
                                                       image[row * width + 2 * index].z * 0.7, image[row * width + 2 * index].w * 0.7);
                uchar4 transparent_pixel_plus_one = make_uchar4(image[row * width + 2 * index + 1].x * 0.7, image[row * width + 2 * index + 1].y * 0.7,
                                                                image[row * width + 2 * index + 1].z * 0.7, image[row * width + 2 * index + 1].w * 0.7);
                image[row * width + 2 * index] = transparent_pixel;
                image[row * width + 2 * index + 1] = transparent_pixel_plus_one;
            }
        }
    }
}

__global__ void rgb2grayKernel(uchar4 *image, uchar4 *outputImage, int height, int width)
{

    int total = width * height;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < total; i += stride)
    {

        unsigned char gray = image[i].x * 0.299 + image[i].y * 0.587 + image[i].z * 0.114;
        outputImage[i].x = gray;
        outputImage[i].y = gray;
        outputImage[i].z = gray;
    }
}

__global__ void calcHistogramKernel(uchar4 *image, int *histogram, int height, int width)
{
    int total = width * height;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int rgbValue;

    __shared__ int histo_local[256];

    if (threadIdx.x < 256)
    {
        histo_local[threadIdx.x] = 0;
    }
    __syncthreads();

    for (size_t i = index; i < total; i += stride)
    {

        rgbValue = image[i].x;
        atomicAdd(&histo_local[rgbValue], 1);
    }
    __syncthreads();
    if (threadIdx.x < 256)
    {
        atomicAdd(&histogram[threadIdx.x], histo_local[threadIdx.x]);
    }
}

int main(int argc, char **argv)
{
    // create input/output streams
    videoSource *input = videoSource::Create(argc, argv, ARG_POSITION(0));
    videoOutput *output = videoOutput::Create(argc, argv, ARG_POSITION(1));

    uchar4 *outputImage = NULL;

    hipMalloc(&outputImage, sizeof(uchar4) * 720 * 1280);

    int hostHistogram[256] = {0};

    int *deviceHistogram = NULL;

    hipMalloc(&deviceHistogram, sizeof(int) * 256);

    if (!input)
        return 0;

    // capture/display loop
    while (true)
    {
        int totalPixels = 0;
        uchar4 *image = NULL;

        //  can be uchar3, uchar4, float3, float4
        int status = 0;                             // see videoSource::Status (OK, TIMEOUT, EOS, ERROR)
        if (!input->Capture(&image, 1000, &status)) // 1000ms timeout (default)
        {
            if (status == videoSource::TIMEOUT)
                continue;
            break; // EOS
        }

        if (output != NULL)
        {
            memset(hostHistogram, 0, sizeof(int) * 256);
            hipMemcpy(deviceHistogram, hostHistogram, 256 * sizeof(int), hipMemcpyHostToDevice);
            rgb2grayKernel<<<16, 1024>>>(image, outputImage, input->GetHeight(), input->GetWidth());
            calcHistogramKernel<<<16, 1024>>>(outputImage, deviceHistogram, input->GetHeight(), input->GetWidth());
            hipMemcpy(hostHistogram, deviceHistogram, 256 * sizeof(int), hipMemcpyDeviceToHost);
            plotHistogramKernel<<<256, 1>>>(outputImage, deviceHistogram, input->GetWidth(), input->GetHeight(), 20000);
            output->Render(outputImage, input->GetWidth(), input->GetHeight());
            // Update status bar
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
                    input->GetHeight(), output->GetFrameRate());

            output->SetStatus(str);
            if (!output->IsStreaming()) // check if the user quit
                break;
            for (int i = 0; i < 256; i++)
            {
                totalPixels += hostHistogram[i];
            }
            printf("%d\n", totalPixels);
        }
    }
    hipFree(outputImage);
    hipFree(deviceHistogram);
}